#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>

void cuda_device_init(void)
    {
    int ndev;
    hipError_t ierr;
    ierr = hipGetDeviceCount(&ndev);
    printf("ndev = %d\n", ndev);
    if(ndev > 16) {
       printf("Error ndev too large.\n");
       return;
    }
    if(ierr != hipSuccess) {
       printf("Error in getting device count.\n");
       return;
    }
    hipDeviceSynchronize();
    printf("There are %d GPUs.\n",ndev);
     
    for(int i=0;i<ndev;i++) {
       hipDeviceProp_t pdev;
       hipGetDeviceProperties(&pdev,i);
       hipDeviceSynchronize();
       printf("Name  : %s\n",pdev.name);
       printf("Capability  : %d %d\n",pdev.major,pdev.minor);
       printf("Memory Global: %d Mb\n",(pdev.totalGlobalMem+1024*1024)/1024/1024);
       printf("Memory Const : %d Kb\n",pdev.totalConstMem/1024);
       printf("Memory Shared: %d Kb\n",pdev.sharedMemPerBlock/1024);
       printf("Clock  : %.3f GHz\n",pdev.clockRate/1000000.0);
       printf("Processors  : %d\n",pdev.multiProcessorCount);
       printf("Cores  : %d\n",8*pdev.multiProcessorCount);
       printf("Warp  : %d\n",pdev.warpSize);
       printf("Max Thr/Blk  : %d\n",pdev.maxThreadsPerBlock);
       printf("Max Blk Size : %d %d %d\n",pdev.maxThreadsDim[0],pdev.maxThreadsDim[1],pdev.maxThreadsDim[2]);
       printf("Max Grid Size: %d %d %d\n",pdev.maxGridSize[0],pdev.maxGridSize[1],pdev.maxGridSize[2]);
    }
}

int main(int argc, char * argv[]) {

   cuda_device_init();
   return 0;
}
